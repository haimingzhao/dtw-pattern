//
// Created by u1590812 on 20/08/16.
//

#include "MatrixCuda.h"

#include <iostream>
#include <hip/hip_runtime.h>

MatrixCuda::MatrixCuda(const std::string datafile): Matrix(datafile){ }

void MatrixCuda::allocate() {

    std::cout << "Cuda allocate" << std::endl;
    // todo copy X Y to device
#ifdef TIME
    hipEvent_t start , stop ;
    hipEventCreate (& start) ;
    hipEventCreate (& stop) ;
    hipEventRecord ( start ) ;
    float milliseconds = 0.0 ;
#endif

#ifdef TIME
    hipEventRecord ( stop ) ;
    hipEventSynchronize ( stop ) ;
    hipEventElapsedTime(&milliseconds, start, stop ) ;
    std::cout << "Matrix, on copySeries, "<< milliseconds << std::endl;
#endif


#ifdef TIME
//    hipEvent_t start , stop ;
    hipEventCreate (& start) ;
    hipEventCreate (& stop) ;
    hipEventRecord ( start ) ;
    milliseconds = 0.0 ;
#endif

    // allocate matrix
    hipMalloc(&C, (nx*ny)*sizeof(double));
    hipMalloc(&D, (nx*ny)*sizeof(double));

    hipMalloc(&L, (nx*ny)*sizeof(size_t));
    hipMalloc(&Rsi, (nx*ny)*sizeof(size_t));
    hipMalloc(&Rsj, (nx*ny)*sizeof(size_t));
    hipMalloc(&Rli, (nx*ny)*sizeof(size_t));
    hipMalloc(&Rlj, (nx*ny)*sizeof(size_t));
    hipMalloc(&Pi, (nx*ny)*sizeof(size_t));
    hipMalloc(&Pj, (nx*ny)*sizeof(size_t));

    hipMalloc(&visited, (nx*ny)*sizeof(bool));
    hipMalloc(&OP, (nx*ny)*sizeof(bool));

#ifdef TIME
    hipEventRecord ( stop ) ;
    hipEventSynchronize ( stop ) ;
    hipEventElapsedTime(&milliseconds, start, stop ) ;
    std::cout << "Matrix, on hipMalloc, "<< milliseconds << std::endl;
#endif

    /* TODO Allocate and initialise anti-diagonal coordinate arrays */

    allocated = true;
}

void MatrixCuda::deallocate() {
#ifdef TIME
    hipEvent_t start , stop ;
    hipEventCreate (& start) ;
    hipEventCreate (& stop) ;
    hipEventRecord ( start ) ;
    float milliseconds = 0.0 ;
#endif
    hipFree(C);
    hipFree(D);
    hipFree(L);
    hipFree(Rsi);
    hipFree(Rsj);
    hipFree(Rli);
    hipFree(Rlj);

    hipFree(Pi);
    hipFree(Pj);

    hipFree(visited);
    hipFree(OP);

#ifdef TIME
    hipEventRecord ( stop ) ;
    hipEventSynchronize ( stop ) ;
    hipEventElapsedTime(&milliseconds, start, stop ) ;
    std::cout << "Matrix, on hipFree, "<< milliseconds << std::endl;
#endif
}

MatrixCuda::~MatrixCuda() {
    deallocate();
}

void MatrixCuda::init() {
    std::cout <<"Cuda init"<< std::endl;
}

void MatrixCuda::dtwm(double t, size_t o) {
    std::cout <<"Cuda dtwm"<< std::endl;
}

void MatrixCuda::findPath(size_t w) {
    std::cout <<"Cuda findPath"<< std::endl;
}


void MatrixCuda::markPath(size_t si, size_t sj, size_t li, size_t lj) {
    std::cout <<"Cuda markPath"<< std::endl;
}
