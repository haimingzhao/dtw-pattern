//
// Created by u1590812 on 20/08/16.
//

#include "MatrixCuda.h"
#include "MatrixKernels.h"
#include <iostream>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

MatrixCuda::MatrixCuda(const std::vector<double> &X, const std::vector<double> &Y): Matrix(X,Y){}

void MatrixCuda::allocate() {
    std::cout << "Cuda allocate" << std::endl;

#ifdef TIME
    hipEvent_t start , stop ;
    hipEventCreate (& start) ;
    hipEventCreate (& stop) ;
    hipEventRecord ( start ) ;
    float milliseconds = 0.0 ;
#endif
    // allocate matrix
    hipMalloc(&I, (nx*ny)*sizeof(size_t));

    hipMalloc(&dX, nx*sizeof(double));
    hipMalloc(&dY, ny*sizeof(double));

    hipMalloc(&C, (nx*ny)*sizeof(double));
    hipMalloc(&D, (nx*ny)*sizeof(double));

    hipMalloc(&L, (nx*ny)*sizeof(size_t));
    hipMalloc(&Rsi, (nx*ny)*sizeof(size_t));
    hipMalloc(&Rsj, (nx*ny)*sizeof(size_t));
    hipMalloc(&Rli, (nx*ny)*sizeof(size_t));
    hipMalloc(&Rlj, (nx*ny)*sizeof(size_t));
    hipMalloc(&Pi, (nx*ny)*sizeof(size_t));
    hipMalloc(&Pj, (nx*ny)*sizeof(size_t));

    hipMalloc(&visited, (nx*ny)*sizeof(bool));
    hipMalloc(&OP, (nx*ny)*sizeof(bool));
#ifdef TIME
    hipEventRecord ( stop ) ;
    hipEventSynchronize ( stop ) ;
    hipEventElapsedTime(&milliseconds, start, stop ) ;
    std::cout << "Matrix, on hipMalloc, "<< milliseconds << std::endl;
#endif


#ifdef TIME
//    hipEvent_t start , stop ;
    hipEventCreate (& start) ;
    hipEventCreate (& stop) ;
    hipEventRecord ( start ) ;
    milliseconds = 0.0 ;
#endif
    // copy X Y to device
    hipMemcpy(dX, X.data(), nx*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(dY, Y.data(), ny*sizeof(double),hipMemcpyHostToDevice);
#ifdef TIME
    hipEventRecord ( stop ) ;
    hipEventSynchronize ( stop ) ;
    hipEventElapsedTime(&milliseconds, start, stop ) ;
    std::cout << "Matrix, on hipMemcpy time series, "<< milliseconds << std::endl;
#endif

    allocated = true;
}

void MatrixCuda::deallocate() {
#ifdef TIME
    hipEvent_t start , stop ;
    hipEventCreate (& start) ;
    hipEventCreate (& stop) ;
    hipEventRecord ( start ) ;
    float milliseconds = 0.0 ;
#endif
    hipFree(C);
    hipFree(D);
    hipFree(L);
    hipFree(Rsi);
    hipFree(Rsj);
    hipFree(Rli);
    hipFree(Rlj);

    hipFree(Pi);
    hipFree(Pj);

    hipFree(visited);
    hipFree(OP);

#ifdef TIME
    hipEventRecord ( stop ) ;
    hipEventSynchronize ( stop ) ;
    hipEventElapsedTime(&milliseconds, start, stop ) ;
    std::cout << "Matrix, on hipFree, "<< milliseconds << std::endl;
#endif
}

MatrixCuda::~MatrixCuda() {
    deallocate();
}

void MatrixCuda::init() {
    std::cout <<"Cuda init"<< std::endl;

    hipMemset(I, 0, (nx*ny)*sizeof(size_t));          // init I to 0 for debug
    // init D in initCUDA since hipMemset only handle bytes
    hipMemset(L, 0, (nx*ny)*sizeof(size_t));          // init L to 0 for empty lengths

    //todo: are these needed?
    hipMemset(Rsi, 0, (nx*ny)*sizeof(size_t));
    hipMemset(Rsj, 0, (nx*ny)*sizeof(size_t));
    hipMemset(Rli, 0, (nx*ny)*sizeof(size_t));
    hipMemset(Rlj, 0, (nx*ny)*sizeof(size_t));
    hipMemset(Pi, 0, (nx*ny)*sizeof(size_t));
    hipMemset(Pj, 0, (nx*ny)*sizeof(size_t));

    hipMemset(visited, 0, (nx*ny)*sizeof(bool)); // init visited to 0 (false)
    hipMemset(OP,      0, (nx*ny)*sizeof(bool)); // init OptimalPath marks to 0 (false)


    // CUDA calculate anti-diagonal index in I and calculate cost in parallel for all cells
    const size_t num_blocks = (nx*ny + BLOCK_SIZE-1)/BLOCK_SIZE; // rounding up dividing by BLOCK_SIZE

    initCuda<<<num_blocks, BLOCK_SIZE>>>(I, C, D, dX, dY, nx, ny);
/*//    size_t idx;
//    for (size_t i = 0; i < nx; ++i) {
//        for (size_t j = 0; j < ny; ++j) {
//            if ( i >= j ){
//                I[i*ny+ j] = getI_bl(i,j, nx, ny);
//            }else{
//                size_t uj0= getI_bl(0,0, nx, ny) ;
//                I[i*ny+ j] = getI_ur(i,j, nx, ny, uj0);
//            }
//            idx = I[i*ny+ j];
//            C[idx] = getCost(i, j);
//        }
//    }

//    std::cout<< "indexes: " << std::endl;
//    for (size_t i = 0; i < nx; ++i) {
//        for (size_t j = 0; j < ny; ++j) {
//            std::cout<< I[i][j] << " " ;
//        }
//        std::cout << std::endl;
//    }
//
//    // C and D matrix initialisation anti-diagonal --not need to
//    size_t idx;
//    for (size_t si = 0; si < nx; ++si) {
//        size_t i = si + 1; // because while loop has i--
//        size_t j = 0 ;
//        while (i-- && j < ny){
//            idx = I[i][j];
//            C[idx] = getCost(i, j);
//            D[idx] = cuda_inf;
//            j = j + 1;
//        }
//    }
//
//    for (size_t sj = 1; sj < ny; ++sj) {
//        size_t i = nx ;  // which is nx = i end index +1, because we need it for i--
//        size_t j = sj ;
//        while (i-- && j < ny){
//            idx = I[i][j];
//            C[idx] = getCost(i, j);
//            D[idx] = cuda_inf;
//            j = j + 1;
//        }
//    }*/
}

void MatrixCuda::dtwm(double t, size_t o) {
    std::cout <<"Cuda dtwm"<< std::endl;

    // run CUDA in parallel in an anti-diagonal strip way
    const size_t num_blocks = (ny + BLOCK_SIZE-1)/BLOCK_SIZE; // rounding up dividing by BLOCK_SIZE

    dtwmCuda<<<num_blocks, BLOCK_SIZE>>>(I, C, D, L, Rsi, Rsj, Rli, Rlj, Pi, Pj, t, o, nx, ny);

/*//    for (size_t si = 0; si < nx; ++si) {
//        size_t i = si + 1; // because while loop has i--
//        size_t j = 0 ;
//        while (i-- && j < ny){
//            dtwm_task(i, j, I, t, o,
//                      C, D, L, Rsi, Rsj, Rli, Rlj, Pi, Pj);
//            j = j + 1;
//        }
//    }
//
//    for (size_t sj = 1; sj < ny; ++sj) {
//        size_t i = nx ;  // which is nx = i end index +1, because we need it for i--
//        size_t j = sj ;
//        while (i-- && j < ny){
//            dtwm_task(i, j, I, t, o,
//                      C, D, L, Rsi, Rsj, Rli, Rlj, Pi, Pj);
//            j = j + 1;
//        }
//    }*/
}

void MatrixCuda::findPath(size_t w) {
    std::cout <<"Cuda findPath"<< std::endl;

    const size_t num_blocks = (nx*ny + BLOCK_SIZE-1)/BLOCK_SIZE; // rounding up dividing by BLOCK_SIZE
    findPathCuda<<<num_blocks, BLOCK_SIZE>>>(I, L, Rli, Rlj, Pi, Pj, OP, w, nx, ny);

/*//    for (size_t si = 0; si < nx; ++si) {
//        size_t i = si + 1; // because while loop has i--
//        size_t j = 0 ;
//        while (i-- && j < ny){
//            findPath_task(i,j, I, ny, w,
//                    L, Rli, Rlj, Pi, Pj, OP);
//            j = j + 1;
//        }
//    }
//
//    for (size_t sj = 1; sj < ny; ++sj) {
//        size_t i = nx ;  // which is nx = i end index +1, because we need it for i--
//        size_t j = sj ;
//        while (i-- && j < ny){
//            findPath_task(i,j, I, ny, w,
//                          L, Rli, Rlj, Pi, Pj, OP);
//            j = j + 1;
//        }
//    }*/
}

double *MatrixCuda::getC() {
    double *hC = new double[nx*ny];
    hipMemcpy(hC, C, nx*ny*sizeof(double), hipMemcpyDeviceToHost);
    return hC;
}

double *MatrixCuda::getD() {
    double *hD = new double[nx*ny];
    hipMemcpy(hD, D, nx*ny*sizeof(double), hipMemcpyDeviceToHost);
    return hD;
}

size_t *MatrixCuda::getL() {
    size_t *hL = new size_t[nx*ny];
    hipMemcpy(hL, L, nx*ny*sizeof(size_t), hipMemcpyDeviceToHost);
    return hL;
}

bool *MatrixCuda::getOP() {
    bool *hOP = new bool[nx*ny];
    hipMemcpy(hOP, OP, nx*ny*sizeof(bool), hipMemcpyDeviceToHost);
    return hOP;     // no need to rearrange since OP is arranged normally
}

size_t *MatrixCuda::getI() {
    size_t *hI = new size_t[nx*ny];
    hipMemcpy(hI, I, nx*ny*sizeof(size_t), hipMemcpyDeviceToHost);
    return hI;     // no need to rearrange since OP is arranged normally
}
