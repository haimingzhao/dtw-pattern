//
// Created by u1590812 on 20/08/16.
//

#include "MatrixCuda.h"
#include "MatrixKernels.h"
#include <iostream>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

MatrixCuda::MatrixCuda(const std::vector<double> &X, const std::vector<double> &Y): Matrix(X,Y){}

void MatrixCuda::allocate() {

#ifdef TIME
    hipEvent_t start , stop ;
    hipEventCreate (& start) ;
    hipEventCreate (& stop) ;
    hipEventRecord ( start ) ;
    float milliseconds = 0.0 ;
#endif
    // allocate matrix
    hipError_t error;
    error = hipMalloc(&I, (nx*ny)*sizeof(size_t)); if( error != hipSuccess ) { std::cerr << "Failed at malloc.\n"; return; }

    error = hipMalloc(&dX, nx*sizeof(double));     if( error != hipSuccess ) { std::cerr << "Failed at malloc.\n"; return; }
    error = hipMalloc(&dY, ny*sizeof(double));     if( error != hipSuccess ) { std::cerr << "Failed at malloc.\n"; return; }

    error = hipMalloc(&C, (nx*ny)*sizeof(double));     if( error != hipSuccess ) { std::cerr << "Failed at malloc.\n"; return; }
    error = hipMalloc(&D, (nx*ny)*sizeof(double));     if( error != hipSuccess ) { std::cerr << "Failed at malloc.\n"; return; }

    error = hipMalloc(&L, (nx*ny)*sizeof(size_t));     if( error != hipSuccess ) { std::cerr << "Failed at malloc.\n"; return; }
    error = hipMalloc(&Rsi, (nx*ny)*sizeof(size_t));   if( error != hipSuccess ) { std::cerr << "Failed at malloc.\n"; return; }
    error = hipMalloc(&Rsj, (nx*ny)*sizeof(size_t));   if( error != hipSuccess ) { std::cerr << "Failed at malloc.\n"; return; }
    error = hipMalloc(&Rli, (nx*ny)*sizeof(size_t));   if( error != hipSuccess ) { std::cerr << "Failed at malloc.\n"; return; }
    error = hipMalloc(&Rlj, (nx*ny)*sizeof(size_t));   if( error != hipSuccess ) { std::cerr << "Failed at malloc.\n"; return; }
    error = hipMalloc(&Pi, (nx*ny)*sizeof(size_t));    if( error != hipSuccess ) { std::cerr << "Failed at malloc.\n"; return; }
    error = hipMalloc(&Pj, (nx*ny)*sizeof(size_t));    if( error != hipSuccess ) { std::cerr << "Failed at malloc.\n"; return; }

    error = hipMalloc(&visited, (nx*ny)*sizeof(bool)); if( error != hipSuccess ) { std::cerr << "Failed at malloc.\n"; return; }
    error = hipMalloc(&OP, (nx*ny)*sizeof(bool));      if( error != hipSuccess ) { std::cerr << "Failed at malloc.\n"; return; }
#ifdef TIME
    hipEventRecord ( stop ) ;
    hipEventSynchronize ( stop ) ;
    hipEventElapsedTime(&milliseconds, start, stop ) ;
    std::cout << getClasstype() << ", on hipMalloc, "<< milliseconds << std::endl;
#endif


#ifdef TIME
//    hipEvent_t start , stop ;
    hipEventCreate (& start) ;
    hipEventCreate (& stop) ;
    hipEventRecord ( start ) ;
    milliseconds = 0.0 ;
#endif
    // copy X Y to device
    hipMemcpy(dX, X.data(), nx*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(dY, Y.data(), ny*sizeof(double),hipMemcpyHostToDevice);
#ifdef TIME
    hipEventRecord ( stop ) ;
    hipEventSynchronize ( stop ) ;
    hipEventElapsedTime(&milliseconds, start, stop ) ;
    std::cout << getClasstype() << ", on hipMemcpy time series, "<< milliseconds << std::endl;
#endif

    allocated = true;
}

void MatrixCuda::deallocate() {
#ifdef TIME
    hipEvent_t start , stop ;
    hipEventCreate (& start) ;
    hipEventCreate (& stop) ;
    hipEventRecord ( start ) ;
    float milliseconds = 0.0 ;
#endif
    hipFree(I);
    hipFree(dX);
    hipFree(dY);

    hipFree(C);
    hipFree(D);
    hipFree(L);
    hipFree(Rsi);
    hipFree(Rsj);
    hipFree(Rli);
    hipFree(Rlj);

    hipFree(Pi);
    hipFree(Pj);

    hipFree(visited);
    hipFree(OP);

#ifdef TIME
    hipEventRecord ( stop ) ;
    hipEventSynchronize ( stop ) ;
    hipEventElapsedTime(&milliseconds, start, stop ) ;
    std::cout << getClasstype()  << ", on hipFree, "<< milliseconds << std::endl;
#endif
}

MatrixCuda::~MatrixCuda() {
    deallocate();
}

void MatrixCuda::init() {

    hipMemset(I, 0, (nx*ny)*sizeof(size_t));          // init I to 0 for debug
    // init D in initCUDA since hipMemset only handle bytes
    hipMemset(L, 0, (nx*ny)*sizeof(size_t));          // init L to 0 for empty lengths

    //todo: are these needed?
    hipMemset(Rsi, 0, (nx*ny)*sizeof(size_t));
    hipMemset(Rsj, 0, (nx*ny)*sizeof(size_t));
    hipMemset(Rli, 0, (nx*ny)*sizeof(size_t));
    hipMemset(Rlj, 0, (nx*ny)*sizeof(size_t));
    hipMemset(Pi, 0, (nx*ny)*sizeof(size_t));
    hipMemset(Pj, 0, (nx*ny)*sizeof(size_t));

    hipMemset(visited, 0, (nx*ny)*sizeof(bool)); // init visited to 0 (false)
    hipMemset(OP,      0, (nx*ny)*sizeof(bool)); // init OptimalPath marks to 0 (false)


    // CUDA calculate anti-diagonal index in I and calculate cost in parallel for all cells
    const size_t num_blocks = (nx*ny + BLOCK_SIZE-1)/BLOCK_SIZE; // rounding up dividing by BLOCK_SIZE

    initCuda<<<num_blocks, BLOCK_SIZE>>>(I, C, D, dX, dY, nx, ny);

}

void MatrixCuda::dtwm(double t, size_t o) {

    // run CUDA in parallel in an anti-diagonal strip way
    const size_t num_blocks = (ny + BLOCK_SIZE-1)/BLOCK_SIZE; // rounding up dividing by BLOCK_SIZE
    dtwmCuda<<<num_blocks, BLOCK_SIZE>>>(I, C, D, L, Rsi, Rsj, Rli, Rlj, Pi, Pj, t, o, nx, ny);

}

void MatrixCuda::findPath(size_t w) {

    const size_t num_blocks = (nx*ny + BLOCK_SIZE-1)/BLOCK_SIZE; // rounding up dividing by BLOCK_SIZE
    findPathCuda<<<num_blocks, BLOCK_SIZE>>>(I, L, Rli, Rlj, Pi, Pj, OP, w, nx, ny);

}

double *MatrixCuda::getC() {
    double *hC = new double[nx*ny];
    hipMemcpy(hC, C, nx*ny*sizeof(double), hipMemcpyDeviceToHost);
    return hC;
}

double *MatrixCuda::getD() {
    double *hD = new double[nx*ny];
    hipMemcpy(hD, D, nx*ny*sizeof(double), hipMemcpyDeviceToHost);
    return hD;
}

size_t *MatrixCuda::getL() {
    size_t *hL = new size_t[nx*ny];
    hipMemcpy(hL, L, nx*ny*sizeof(size_t), hipMemcpyDeviceToHost);
    return hL;
}

bool *MatrixCuda::getOP() {
    bool *hOP = new bool[nx*ny];
    hipMemcpy(hOP, OP, nx*ny*sizeof(bool), hipMemcpyDeviceToHost);
    return hOP;     // no need to rearrange since OP is arranged normally
}

size_t *MatrixCuda::getI() {
    size_t *hI = new size_t[nx*ny];
    hipMemcpy(hI, I, nx*ny*sizeof(size_t), hipMemcpyDeviceToHost);
    return hI;     // no need to rearrange since OP is arranged normally
}
