#include "hip/hip_runtime.h"
//
// Created by u1590812 on 20/08/16.
//

#include "MatrixCudaOp.h"
#include "MatrixKernels.h"
#include <iostream>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

// infinity value for host to copy to cuda
//#define inf DBL_MAX
//double inf = (std::numeric_limits<double>::max());
//#define cuda_inf HIP_INF

MatrixCudaOp::MatrixCudaOp(const std::string datafile): Matrix(datafile){

//    // Allocate and initialise 2D diagonal index matrix - move to cuda
//    I = new size_t*[nx];
//    for (size_t i = 0; i < nx; ++i) {
//        I[i] = new size_t[ny]();
//    }

    /* Initialise anti-diagonal memory location coordinates using while loop
     * i is row -> X of length nx,
     * j is column -> Y of length ny
     * careful for size_t being unsigned */
//    size_t idx = 0;
//    for (size_t si = nx; si--; ) {
//        size_t i = si;
//        size_t j = 0 ;
//        while (i < nx && j < ny){
//            I[i][j] = idx;
//            ++ idx;
//            i = i + 1;
//            j = j + 1;
//        }
//    }
//    for (size_t sj = 1; sj < ny; ++sj) {
//        size_t i =  0;
//        size_t j = sj;
//        while (i < nx && j < ny){
//            I[i][j] = idx;
//            ++ idx;
//            i = i + 1;
//            j = j + 1;
//        }
//    }

}

void MatrixCudaOp::allocate() {
    std::cout << "Cuda allocate" << std::endl;

#ifdef TIME
    hipEvent_t start , stop ;
    hipEventCreate (& start) ;
    hipEventCreate (& stop) ;
    hipEventRecord ( start ) ;
    float milliseconds = 0.0 ;
#endif
    // allocate matrix
    hipMalloc(&I, (nx*ny)*sizeof(size_t));

    hipMalloc(&dX, nx*sizeof(double));
    hipMalloc(&dY, ny*sizeof(double));

    hipMalloc(&C, (nx*ny)*sizeof(double));
    hipMalloc(&D, (nx*ny)*sizeof(double));

    hipMalloc(&L, (nx*ny)*sizeof(size_t));
    hipMalloc(&Rsi, (nx*ny)*sizeof(size_t));
    hipMalloc(&Rsj, (nx*ny)*sizeof(size_t));
    hipMalloc(&Rli, (nx*ny)*sizeof(size_t));
    hipMalloc(&Rlj, (nx*ny)*sizeof(size_t));
    hipMalloc(&Pi, (nx*ny)*sizeof(size_t));
    hipMalloc(&Pj, (nx*ny)*sizeof(size_t));

    hipMalloc(&visited, (nx*ny)*sizeof(bool));
    hipMalloc(&OP, (nx*ny)*sizeof(bool));
#ifdef TIME
    hipEventRecord ( stop ) ;
    hipEventSynchronize ( stop ) ;
    hipEventElapsedTime(&milliseconds, start, stop ) ;
    std::cout << "Matrix, on hipMalloc, "<< milliseconds << std::endl;
#endif


#ifdef TIME
//    hipEvent_t start , stop ;
    hipEventCreate (& start) ;
    hipEventCreate (& stop) ;
    hipEventRecord ( start ) ;
    milliseconds = 0.0 ;
#endif
    // copy X Y to device
    hipMemcpy(dX, X.data(), nx*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(dY, Y.data(), ny*sizeof(double),hipMemcpyHostToDevice);
#ifdef TIME
    hipEventRecord ( stop ) ;
    hipEventSynchronize ( stop ) ;
    hipEventElapsedTime(&milliseconds, start, stop ) ;
    std::cout << "Matrix, on hipMemcpy time series, "<< milliseconds << std::endl;
#endif

    allocated = true;
}

void MatrixCudaOp::deallocate() {
#ifdef TIME
    hipEvent_t start , stop ;
    hipEventCreate (& start) ;
    hipEventCreate (& stop) ;
    hipEventRecord ( start ) ;
    float milliseconds = 0.0 ;
#endif
    hipFree(C);
    hipFree(D);
    hipFree(L);
    hipFree(Rsi);
    hipFree(Rsj);
    hipFree(Rli);
    hipFree(Rlj);

    hipFree(Pi);
    hipFree(Pj);

    hipFree(visited);
    hipFree(OP);

#ifdef TIME
    hipEventRecord ( stop ) ;
    hipEventSynchronize ( stop ) ;
    hipEventElapsedTime(&milliseconds, start, stop ) ;
    std::cout << "Matrix, on hipFree, "<< milliseconds << std::endl;
#endif
}

MatrixCudaOp::~MatrixCudaOp() {
    deallocate();
}

void MatrixCudaOp::MatrixCudaOp::init() {
    std::cout <<"Cuda init"<< std::endl;

    hipMemset(I, 0, (nx*ny)*sizeof(size_t));          // init I to 0 for debug
    // init D in initCUDA since hipMemset only handle bytes
    hipMemset(L, 0, (nx*ny)*sizeof(size_t));          // init L to 0 for empty lengths

    //todo: are these needed?
    hipMemset(Rsi, 0, (nx*ny)*sizeof(size_t));
    hipMemset(Rsj, 0, (nx*ny)*sizeof(size_t));
    hipMemset(Rli, 0, (nx*ny)*sizeof(size_t));
    hipMemset(Rlj, 0, (nx*ny)*sizeof(size_t));
    hipMemset(Pi, 0, (nx*ny)*sizeof(size_t));
    hipMemset(Pj, 0, (nx*ny)*sizeof(size_t));

    hipMemset(visited, 0, (nx*ny)*sizeof(bool)); // init visited to 0 (false)
    hipMemset(OP,      0, (nx*ny)*sizeof(bool)); // init OptimalPath marks to 0 (false)


    // CUDA calculate anti-diagonal index in I and calculate cost in parallel for all cells
    const size_t num_blocks = (nx*ny + BLOCK_SIZE-1)/BLOCK_SIZE; // rounding up dividing by BLOCK_SIZE

    initCudaOp<<<num_blocks, BLOCK_SIZE>>>(I, C, D, dX, dY, nx, ny);
/*//    size_t idx;
//    for (size_t i = 0; i < nx; ++i) {
//        for (size_t j = 0; j < ny; ++j) {
//            if ( i >= j ){
//                I[i*ny+ j] = getI_bl(i,j, nx, ny);
//            }else{
//                size_t uj0= getI_bl(0,0, nx, ny) ;
//                I[i*ny+ j] = getI_ur(i,j, nx, ny, uj0);
//            }
//            idx = I[i*ny+ j];
//            C[idx] = getCost(i, j);
//        }
//    }

//    std::cout<< "indexes: " << std::endl;
//    for (size_t i = 0; i < nx; ++i) {
//        for (size_t j = 0; j < ny; ++j) {
//            std::cout<< I[i][j] << " " ;
//        }
//        std::cout << std::endl;
//    }
//
//    // C and D matrix initialisation anti-diagonal --not need to
//    size_t idx;
//    for (size_t si = 0; si < nx; ++si) {
//        size_t i = si + 1; // because while loop has i--
//        size_t j = 0 ;
//        while (i-- && j < ny){
//            idx = I[i][j];
//            C[idx] = getCost(i, j);
//            D[idx] = cuda_inf;
//            j = j + 1;
//        }
//    }
//
//    for (size_t sj = 1; sj < ny; ++sj) {
//        size_t i = nx ;  // which is nx = i end index +1, because we need it for i--
//        size_t j = sj ;
//        while (i-- && j < ny){
//            idx = I[i][j];
//            C[idx] = getCost(i, j);
//            D[idx] = cuda_inf;
//            j = j + 1;
//        }
//    }*/
}

void MatrixCudaOp::dtwm(double t, size_t o) {
    std::cout <<"Cuda dtwm"<< std::endl;

    // run CUDA in parallel in an anti-diagonal strip way
    const size_t num_blocks = (ny + BLOCK_SIZE-1)/BLOCK_SIZE; // rounding up dividing by BLOCK_SIZE

    dtwmCuda<<<num_blocks, BLOCK_SIZE>>>(I, C, D, L, Rsi, Rsj, Rli, Rlj, Pi, Pj, t, o, nx, ny);

/*//    for (size_t si = 0; si < nx; ++si) {
//        size_t i = si + 1; // because while loop has i--
//        size_t j = 0 ;
//        while (i-- && j < ny){
//            dtwm_task(i, j, I, t, o,
//                      C, D, L, Rsi, Rsj, Rli, Rlj, Pi, Pj);
//            j = j + 1;
//        }
//    }
//
//    for (size_t sj = 1; sj < ny; ++sj) {
//        size_t i = nx ;  // which is nx = i end index +1, because we need it for i--
//        size_t j = sj ;
//        while (i-- && j < ny){
//            dtwm_task(i, j, I, t, o,
//                      C, D, L, Rsi, Rsj, Rli, Rlj, Pi, Pj);
//            j = j + 1;
//        }
//    }*/
}

void MatrixCudaOp::findPath(size_t w) {
    std::cout <<"Cuda findPath"<< std::endl;

    const size_t num_blocks = (nx*ny + BLOCK_SIZE-1)/BLOCK_SIZE; // rounding up dividing by BLOCK_SIZE
    findPathCuda<<<num_blocks, BLOCK_SIZE>>>(I, L, Rli, Rlj, Pi, Pj, OP, w, nx, ny);

/*//    for (size_t si = 0; si < nx; ++si) {
//        size_t i = si + 1; // because while loop has i--
//        size_t j = 0 ;
//        while (i-- && j < ny){
//            findPath_task(i,j, I, ny, w,
//                    L, Rli, Rlj, Pi, Pj, OP);
//            j = j + 1;
//        }
//    }
//
//    for (size_t sj = 1; sj < ny; ++sj) {
//        size_t i = nx ;  // which is nx = i end index +1, because we need it for i--
//        size_t j = sj ;
//        while (i-- && j < ny){
//            findPath_task(i,j, I, ny, w,
//                          L, Rli, Rlj, Pi, Pj, OP);
//            j = j + 1;
//        }
//    }*/
}

double *MatrixCudaOp::getC() {
    double *hC = new double[nx*ny];
    double *hCn = new double[nx*ny];
    hipMemcpy(hC, C, nx*ny*sizeof(double), hipMemcpyDeviceToHost);
    // copy host copy to a normal index arrangement
    size_t idx = 0;
    for (size_t si = nx; si--; ) {
        size_t i = si;
        size_t j = 0 ;
        while (i < nx && j < ny){
            hCn[i*ny +j] = hC[idx];

            ++ idx;
            i = i + 1;
            j = j + 1;
        }
    }
    for (size_t sj = 1; sj < ny; ++sj) {
        size_t i =  0;
        size_t j = sj;
        while (i < nx && j < ny){
            hCn[i*ny +j] = hC[idx];

            ++ idx;
            i = i + 1;
            j = j + 1;
        }
    }
    return hCn;
}

double *MatrixCudaOp::getD() {
    double *hD = new double[nx*ny];
    double *hDn = new double[nx*ny];
    hipMemcpy(hD, D, nx*ny*sizeof(double), hipMemcpyDeviceToHost);
    // copy host copy to a normal index arrangement
    size_t idx = 0;
    for (size_t si = nx; si--; ) {
        size_t i = si;
        size_t j = 0 ;
        while (i < nx && j < ny){
            hDn[i*ny +j] = hD[idx];

            ++ idx;
            i = i + 1;
            j = j + 1;
        }
    }
    for (size_t sj = 1; sj < ny; ++sj) {
        size_t i =  0;
        size_t j = sj;
        while (i < nx && j < ny){
            hDn[i*ny +j] = hD[idx];

            ++ idx;
            i = i + 1;
            j = j + 1;
        }
    }
    return hDn;
}

size_t *MatrixCudaOp::getL() {
    size_t *hL = new size_t[nx*ny];
    size_t *hLn = new size_t[nx*ny];
    hipMemcpy(hL, L, nx*ny*sizeof(size_t), hipMemcpyDeviceToHost);
    // copy host copy to a normal index arrangement
    size_t idx = 0;
    for (size_t si = nx; si--; ) {
        size_t i = si;
        size_t j = 0 ;
        while (i < nx && j < ny){
            hLn[i*ny +j] = hL[idx];

            ++ idx;
            i = i + 1;
            j = j + 1;
        }
    }
    for (size_t sj = 1; sj < ny; ++sj) {
        size_t i =  0;
        size_t j = sj;
        while (i < nx && j < ny){
            hLn[i*ny +j] = hL[idx];

            ++ idx;
            i = i + 1;
            j = j + 1;
        }
    }
    return hLn;
}

bool *MatrixCudaOp::getOP() {
    bool *hOP = new bool[nx*ny];
    hipMemcpy(hOP, OP, nx*ny*sizeof(bool), hipMemcpyDeviceToHost);
    return hOP;     // no need to rearrange since OP is arranged normally
}

size_t *MatrixCudaOp::getI() {
    size_t *hI = new size_t[nx*ny];
    hipMemcpy(hI, I, nx*ny*sizeof(size_t), hipMemcpyDeviceToHost);
    return hI;     // no need to rearrange since OP is arranged normally
}
